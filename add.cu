//must include cuda runtime to initial cuda device
#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
//cuda kernel function
__global__ void add_kernel(float *result, float *operand_A, float *operand_B)
{
    int i=threadIdx.x;
    result[i]=operand_A[i]+operand_B[i];
}

__global__ void init_array(float *operand_A,float *operand_B)
{
    int i=threadIdx.x;
    operand_A[i]=2.0f;
    operand_B[i]=5.0f;
}
// __global__ void init_array3(float operand_A,float *c)
// {
//     int i=threadIdx.x;
//     if(operand_A>2.0)
//         *c=operand_A;
//     else
//         *c=1.0;
// }

int main()
{   
    int array_size=10;
    float *a,*b,*x;
    //allocate unified memory
    hipMallocManaged(&a,array_size*sizeof(float));
    hipMallocManaged(&b,array_size*sizeof(float));
    hipMallocManaged(&x,array_size*sizeof(float));
    //run kernel
    //platform("./file_list.txt")
    //platform.sim(string func_name,class cta,p1,p2)
    init_array<<<1,array_size>>>(a,b);
    add_kernel<<<1,array_size>>>(x,a,b);
    //wait for GPU
    hipDeviceSynchronize();
    //Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < array_size; i++)
        maxError = fmax(maxError, fabs(x[i]-2.0f));
    std::cout << "done "<< maxError<<std::endl;
    //free memory
    hipFree(x);
    hipFree(a);
    hipFree(b);

}