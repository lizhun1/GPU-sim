//must include cuda runtime to initial cuda device
#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
//cuda kernel function
__global__ void add_kernel(double *result, double *operand_A, double *operand_B)
{
    int i=threadIdx.x;
    result[i]=operand_A[i]+operand_B[i];
}

__global__ void init_array(double *operand_A,double *operand_B)
{
    int i=threadIdx.x;
    operand_A[i]=2.0f;
    operand_B[i]=5.0f;
}
__global__ void init_array_setp(double *operand_A,double *operand_B)
{
    int i=threadIdx.x;
    if(i<16){
        operand_A[i]=2.0f;
        operand_B[i]=5.0f;
    }
    else{
        operand_A[i]=0;
        operand_B[i]=0;
    }
    
}
__global__ void init_array_bra(double *operand_A,double *operand_B)
{
    int i=threadIdx.x;
    double a=1.0f;
    double b=2.0f;
    if(i==0){
       
        a=a+1;
        b=b+3;
        operand_A[i]=a;
        operand_B[i]=b;
    }
    else{
        b=b-4;
        a=a-2;
        operand_A[i]=a;
        operand_B[i]=b;

    }
    operand_A[i]=operand_A[i]+a;
    operand_B[i]=operand_B[i]+b;
    
}
// __global__ void init_array3(float operand_A,float *c)
// {
//     int i=threadIdx.x;
//     if(operand_A>2.0)
//         *c=operand_A;
//     else
//         *c=1.0;
// }

int main()
{   
    int array_size=10;
    double *a,*b,*x;
    //allocate unified memory
    hipMallocManaged(&a,array_size*sizeof(double));
    hipMallocManaged(&b,array_size*sizeof(double));
    hipMallocManaged(&x,array_size*sizeof(double));
    //run kernel
    //platform("./file_list.txt")
    //platform.sim(string func_name,class cta,p1,p2)
    init_array<<<1,array_size>>>(a,b);
    add_kernel<<<1,array_size>>>(x,a,b);
    //wait for GPU
    hipDeviceSynchronize();
    //Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < array_size; i++)
        maxError = fmax(maxError, fabs(x[i]-2.0f));
    std::cout << "done "<< maxError<<std::endl;
    //free memory
    hipFree(x);
    hipFree(a);
    hipFree(b);

}