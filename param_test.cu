//must include cuda runtime to initial cuda device
#include "hip/hip_runtime.h"
#include "stdio.h"
#include <iostream>
//cuda kernel function
class a_class{
    public:
        int b=1;
        int d=2;
        double f=1.0;
        a_class(){};
        ~a_class(){};
        int get()
        {
            return b;};
};
__global__ void add_kernel(float *result, float *operand_A, float *operand_B)
{
    int i=threadIdx.x;
    result[i]=operand_A[i]+operand_B[i];
}

__global__ void init_array(float *operand_A,float *operand_B,a_class c)
{
    int i=threadIdx.x;
    operand_A[i]=1.0f;
    operand_B[i]=c.f;
}
// __global__ void init_array3(float operand_A,float *c)
// {
//     int i=threadIdx.x;
//     if(operand_A>2.0)
//         *c=operand_A;
//     else
//         *c=1.0;
// }

int main()
{   
    int array_size=10;
    float *a,*b,*x;
    a_class a_param;
    //allocate unified memory
    hipMallocManaged(&a,array_size*sizeof(float));
    hipMallocManaged(&b,array_size*sizeof(float));
    hipMallocManaged(&x,array_size*sizeof(float));
    //run kernel
    //platform("./file_list.txt")
    //platform.sim(string func_name,class cta,p1,p2)
    init_array<<<1,array_size>>>(a,b,a_param);
    add_kernel<<<1,array_size>>>(x,a,b);
    //wait for GPU
    hipDeviceSynchronize();
    //Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < array_size; i++)
        maxError = fmax(maxError, fabs(x[i]-6.0f));
    std::cout << "done "<< maxError<<std::endl;
    //free memory
    hipFree(x);
    hipFree(a);
    hipFree(b);

}